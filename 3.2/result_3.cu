#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define BLOCK_SIZE 16

#define M 100
#define N 100
#define K 100
__managed__ int a[M * N];
__managed__ int b[N * K];
__managed__ int c[M * K];

__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{

    int *h_cc = (int*)malloc(sizeof(int)*M*K);
    
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));


    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            a[i * N + j] = rand() % 1024;
        }
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < K; ++j) {
            b[i * K + j] = rand() % 1024;
        }
    }

    unsigned int grid_rows = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    CHECK(hipEventRecord(start));
    //hipEventQuery(start);
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(a, b, c, M, N, K);    
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Time = %g ms.\n", elapsed_time);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    //hipDeviceSynchronize();
    

    cpu_matrix_mult(a, b, h_cc, M, N, K);

    int ok = 1;
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < K; ++j)
        {
            if(fabs(h_cc[i*K + j] - c[i*K + j])>(1.0e-10))
            {
                
                ok = 0;
            }
        }
    }

    if(ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }

    return 0;
}