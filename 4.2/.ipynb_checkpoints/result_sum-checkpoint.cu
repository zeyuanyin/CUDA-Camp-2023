
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 1000000
#define BLOCK_SIZE 256
#define GRID_SIZE 256

__managed__ int source[N];
__managed__ int gpu_result[1] = {0};
__managed__ int gpu_1_pass_result[GRID_SIZE] = {0};
__managed__ int gpu_2_pass_result[1] = {0};
__managed__ int gpu_atomic_result[1] = {0};
__managed__ int gpu_warp_shuffle_result[1] = {0};



__global__ void sum_gpu_naive(int *in, int count, int *out)
{
    int tmp = 0;
    for(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += blockDim.x * gridDim.x)
    {
        atomicAdd(out, in[idx]);
    }
}

__global__ void _shared_2pass_sum_gpu(int *in, int count, int *out)
{
    __shared__ int ken[BLOCK_SIZE];
    //grid_loop
    int shared_tmp=0;
    for(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += blockDim.x * gridDim.x)
    {
        shared_tmp +=in[idx];
    }
    ken[threadIdx.x] = shared_tmp;
    __syncthreads();

    for(int total_threads = BLOCK_SIZE/2; total_threads>=1; total_threads/=2)
    {
        if(threadIdx.x < total_threads)
        {
            ken[threadIdx.x] = ken[threadIdx.x] + ken[threadIdx.x + total_threads]; 
        }
        __syncthreads();
    }

    // block_sum -> share memory[0]
    if(blockIdx.x * blockDim.x < count)
    {
        if(threadIdx.x == 0)
        {
            out[blockIdx.x] = ken[0];
            // memory space wmr
        }
    }
}

__global__ void _shared_atomic_sum_gpu(int *in, int count, int *out)
{
    __shared__ int ken[BLOCK_SIZE];
    //grid_loop
    int shared_tmp=0;
    for(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += blockDim.x * gridDim.x)
    {
        shared_tmp +=in[idx];
    }
    ken[threadIdx.x] = shared_tmp;
    __syncthreads();

    for(int total_threads = BLOCK_SIZE/2; total_threads>=1; total_threads/=2)
    {
        if(threadIdx.x < total_threads)
        {
            ken[threadIdx.x] = ken[threadIdx.x] + ken[threadIdx.x + total_threads]; 
        }
        __syncthreads();
    }
    // block_sum -> share memory[0]
    if(blockIdx.x * blockDim.x < count)
    {
        if(threadIdx.x == 0)
        {
            atomicAdd(out, ken[0]);
        }
    }
}

__global__ void _shared_atomic_shuffle_sum_gpu(int *in, int count, int *out)
{
    __shared__ int ken[BLOCK_SIZE];
    //grid_loop
    int shared_tmp=0;
    for(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += blockDim.x * gridDim.x)
    {
        shared_tmp +=in[idx];
    }
    ken[threadIdx.x] = shared_tmp;
    __syncthreads();

    for(int total_threads = BLOCK_SIZE/2; total_threads>=32; total_threads/=2)
    {
        if(threadIdx.x < total_threads)
        {
            ken[threadIdx.x] = ken[threadIdx.x] + ken[threadIdx.x + total_threads]; 
        }
        __syncthreads();
    }
    int val = ken[threadIdx.x];
    for (int offset = 16; offset > 0; offset /= 2)
    {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }

    // block_sum -> share memory[0]
    if(blockIdx.x * blockDim.x < count)
    {
        if(threadIdx.x == 0)
        {
            atomicAdd(out, val);
        }
    }
}

int main()
{
    int cpu_result =0;


    printf("Init input source[N]\n");
    for(int i =0; i<N; i++)
    {
        source[i] = rand()%10;
    }

    hipEvent_t start, stop_cpu, stop_gpu_naive, stop_gpu_2pass, stop_gpu_atomic, stop_gpu_shuffle;
    hipEventCreate(&start);
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu_naive);
    hipEventCreate(&stop_gpu_2pass);
    hipEventCreate(&stop_gpu_atomic);
    hipEventCreate(&stop_gpu_shuffle);

    hipEventRecord(start);

    for(int i = 0; i<20; i++)
    {
        gpu_result[0] = 0;
        sum_gpu_naive<<<GRID_SIZE, BLOCK_SIZE>>>(source, N, gpu_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_gpu_naive);
    hipEventSynchronize(stop_gpu_naive);



    _shared_2pass_sum_gpu<<<GRID_SIZE, BLOCK_SIZE>>>(source, N, gpu_1_pass_result);
    _shared_2pass_sum_gpu<<<1, BLOCK_SIZE>>>(gpu_1_pass_result, GRID_SIZE, gpu_2_pass_result);
    hipDeviceSynchronize();
    
    hipEventRecord(stop_gpu_2pass);
    hipEventSynchronize(stop_gpu_2pass);


    for(int i = 0; i<20; i++)
    {
        gpu_atomic_result[0] = 0;
        _shared_atomic_sum_gpu<<<GRID_SIZE, BLOCK_SIZE>>>(source, N, gpu_atomic_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_gpu_atomic);
    hipEventSynchronize(stop_gpu_atomic);


    for(int i = 0; i<20; i++)
    {
        gpu_warp_shuffle_result[0] = 0;
        _shared_atomic_shuffle_sum_gpu<<<GRID_SIZE, BLOCK_SIZE>>>(source, N, gpu_warp_shuffle_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_gpu_shuffle);
    hipEventSynchronize(stop_gpu_shuffle);

    for(int i =0; i<N; i++)
    {
        cpu_result +=source[i];
    }

    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);



    float time_cpu, time_gpu_naive, time_gpu_2pass, time_gpu_atomic, time_gpu_shuffle;

    hipEventElapsedTime(&time_gpu_naive, start, stop_gpu_naive);
    hipEventElapsedTime(&time_gpu_2pass, stop_gpu_naive, stop_gpu_2pass);
    hipEventElapsedTime(&time_gpu_atomic, stop_gpu_2pass, stop_gpu_atomic);
    hipEventElapsedTime(&time_gpu_shuffle, stop_gpu_atomic, stop_gpu_shuffle);
    hipEventElapsedTime(&time_cpu, stop_gpu_shuffle, stop_cpu);
    

    printf("CPU time: %.2f\nGPU naive time: %.2f\nGPU 2Pass time: %.2f\nGPU atomic time: %.2f\nGPU shuffle time: %.2f\n",time_cpu, time_gpu_naive/20, time_gpu_2pass, time_gpu_atomic/20, time_gpu_shuffle/20);


    printf("naive Result: %s\nGPU_naive_result: %d;\nCPU_result: %d;\n\n", (gpu_result[0] == cpu_result)?"Pass":"Error", gpu_result[0], cpu_result);
    printf("2Pass Result: %s\nGPU_2Pass_result: %d;\nCPU_result: %d;\n\n", (gpu_2_pass_result[0] == cpu_result)?"Pass":"Error", gpu_2_pass_result[0], cpu_result);
    printf("atomic Result: %s\nGPU_atomic_result: %d;\nCPU_result: %d;\n\n", (gpu_atomic_result[0] == cpu_result)?"Pass":"Error", gpu_atomic_result[0], cpu_result);
    printf("shuffle Result: %s\nGPU_shuffle_result: %d;\nCPU_result: %d;\n", (gpu_warp_shuffle_result[0] == cpu_result)?"Pass":"Error", gpu_warp_shuffle_result[0], cpu_result);
    
    return 0;
}

